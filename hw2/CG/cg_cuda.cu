#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "globals.h"
#include "randdp.h"
#include "timers.h"

//---------------------------------------------------------------------
/* common / main_int_mem / */
static int colidx[NZ];
static int rowstr[NA+1];
static int iv[NA];
static int arow[NA];
static int acol[NAZ];

/* common / main_flt_mem / */
static double aelt[NAZ];
static double a[NZ];
static double x[NA+2];
static double z[NA+2];
static double p[NA+2];
static double q[NA+2];
static double r[NA+2];

/* gpu memory */
static int *gpu_colidx;
static int *gpu_rowstr;
static double *gpu_a;
static double *gpu_x;
static double *gpu_z;
static double *gpu_p;
static double *gpu_q;
static double *gpu_r;
static double *gpu_sum, cpu_sum[NA+2];

/* common / partit_size / */
static int naa;
static int nzz;
static int firstrow;
static int lastrow;
static int firstcol;
static int lastcol;

/* common /urando/ */
static double amult;
static double tran;

/* common /timers/ */
static logical timeron;
//---------------------------------------------------------------------


//---------------------------------------------------------------------
static void conj_grad(int colidx[],
                      int rowstr[],
                      double x[],
                      double z[],
                      double a[],
                      double p[],
                      double q[],
                      double r[],
                      double *rnorm);
static void makea(int n,
                  int nz,
                  double a[],
                  int colidx[],
                  int rowstr[],
                  int firstrow,
                  int lastrow,
                  int firstcol,
                  int lastcol,
                  int arow[],
                  int acol[][NONZER+1],
                  double aelt[][NONZER+1],
                  int iv[]);
static void sparse(double a[],
                   int colidx[],
                   int rowstr[],
                   int n,
                   int nz,
                   int nozer,
                   int arow[],
                   int acol[][NONZER+1],
                   double aelt[][NONZER+1],
                   int firstrow,
                   int lastrow,
                   int nzloc[],
                   double rcond,
                   double shift);
static void sprnvc(int n, int nz, int nn1, double v[], int iv[]);
static int icnvrt(double x, int ipwr2);
static void vecset(int n, double v[], int iv[], int *nzv, int i, double val);
//---------------------------------------------------------------------


int main(int argc, char *argv[])
{
  int i, j, k, it;

  double zeta;
  double rnorm;
  double norm_temp1, norm_temp2;

  double t, mflops, tmax;
  //char Class;
  logical verified;
  double zeta_verify_value, epsilon, err;

  char *t_names[T_last];

  for (i = 0; i < T_last; i++) {
    timer_clear(i);
  }
  
  timer_start(T_init);

  firstrow = 0;
  lastrow  = NA-1;
  firstcol = 0;
  lastcol  = NA-1;

  zeta_verify_value = VALID_RESULT;
  
  printf("\nCG start...\n\n");
  printf(" Size: %11d\n", NA);
  printf(" Iterations: %5d\n", NITER);
  printf("\n");

  naa = NA;
  nzz = NZ;

  //---------------------------------------------------------------------
  // Inialize random number generator
  //---------------------------------------------------------------------
  tran    = 314159265.0;
  amult   = 1220703125.0;
  zeta    = randlc(&tran, amult);

  //---------------------------------------------------------------------
  //  
  //---------------------------------------------------------------------
  makea(naa, nzz, a, colidx, rowstr, 
        firstrow, lastrow, firstcol, lastcol, 
        arow, 
        (int (*)[NONZER+1])(void*)acol, 
        (double (*)[NONZER+1])(void*)aelt,
        iv);

  //---------------------------------------------------------------------
  // Note: as a result of the above call to makea:
  //      values of j used in indexing rowstr go from 0 --> lastrow-firstrow
  //      values of colidx which are col indexes go from firstcol --> lastcol
  //      So:
  //      Shift the col index vals from actual (firstcol --> lastcol ) 
  //      to local, i.e., (0 --> lastcol-firstcol)
  //---------------------------------------------------------------------
  for (j = 0; j < lastrow - firstrow + 1; j++) {
    for (k = rowstr[j]; k < rowstr[j+1]; k++) {
      colidx[k] = colidx[k] - firstcol;
    }
  }

  // move data to gpu
  hipMalloc(&gpu_colidx, sizeof(colidx));
  hipMalloc(&gpu_rowstr, sizeof(rowstr));
  hipMalloc(&gpu_a, sizeof(a));
  hipMalloc(&gpu_x, sizeof(x));
  hipMalloc(&gpu_z, sizeof(z));
  hipMalloc(&gpu_p, sizeof(p));
  hipMalloc(&gpu_q, sizeof(q));
  hipMalloc(&gpu_r, sizeof(r));
  hipMalloc(&gpu_sum, sizeof(cpu_sum));
  hipMemcpy(gpu_colidx, colidx, sizeof(colidx), hipMemcpyHostToDevice);
  hipMemcpy(gpu_rowstr, rowstr, sizeof(rowstr), hipMemcpyHostToDevice);
  hipMemcpy(gpu_a, a, sizeof(a), hipMemcpyHostToDevice);

  //---------------------------------------------------------------------
  // set starting vector to (1, 1, .... 1)
  //---------------------------------------------------------------------
  for (i = 0; i < NA+1; i++) {
    x[i] = 1.0;
  }
  for (j = 0; j < lastcol - firstcol + 1; j++) {
    q[j] = 0.0;
    z[j] = 0.0;
    r[j] = 0.0;
    p[j] = 0.0;
  }

  zeta = 0.0;

  //---------------------------------------------------------------------
  //---->
  // Do one iteration untimed to init all code and data page tables
  //---->                    (then reinit, start timing, to niter its)
  //---------------------------------------------------------------------
  for (it = 1; it <= 1; it++) {
    //---------------------------------------------------------------------
    // The call to the conjugate gradient routine:
    //---------------------------------------------------------------------
    conj_grad(colidx, rowstr, x, z, a, p, q, r, &rnorm);

    //---------------------------------------------------------------------
    // zeta = shift + 1/(x.z)
    // So, first: (x.z)
    // Also, find norm of z
    // So, first: (z.z)
    //---------------------------------------------------------------------
    norm_temp1 = 0.0;
    norm_temp2 = 0.0;
    for (j = 0; j < lastcol - firstcol + 1; j++) {
      norm_temp1 = norm_temp1 + x[j] * z[j];
      norm_temp2 = norm_temp2 + z[j] * z[j];
    }

    norm_temp2 = 1.0 / sqrt(norm_temp2);

    //---------------------------------------------------------------------
    // Normalize z to obtain x
    //---------------------------------------------------------------------
    for (j = 0; j < lastcol - firstcol + 1; j++) {     
      x[j] = norm_temp2 * z[j];
    }
  } // end of do one iteration untimed


  //---------------------------------------------------------------------
  // set starting vector to (1, 1, .... 1)
  //---------------------------------------------------------------------
  for (i = 0; i < NA+1; i++) {
    x[i] = 1.0;
  }

  zeta = 0.0;

  timer_stop(T_init);

  printf(" Initialization time = %15.3f seconds\n", timer_read(T_init));

  timer_start(T_bench);

  //---------------------------------------------------------------------
  //---->
  // Main Iteration for inverse power method
  //---->
  //---------------------------------------------------------------------
  for (it = 1; it <= NITER; it++) {
    //---------------------------------------------------------------------
    // The call to the conjugate gradient routine:
    //---------------------------------------------------------------------
    if (timeron) timer_start(T_conj_grad);
    conj_grad(colidx, rowstr, x, z, a, p, q, r, &rnorm);
    if (timeron) timer_stop(T_conj_grad);

    //---------------------------------------------------------------------
    // zeta = shift + 1/(x.z)
    // So, first: (x.z)
    // Also, find norm of z
    // So, first: (z.z)
    //---------------------------------------------------------------------
    norm_temp1 = 0.0;
    norm_temp2 = 0.0;
    for (j = 0; j < lastcol - firstcol + 1; j++) {
      norm_temp1 = norm_temp1 + x[j]*z[j];
      norm_temp2 = norm_temp2 + z[j]*z[j];
    }

    norm_temp2 = 1.0 / sqrt(norm_temp2);

    zeta = SHIFT + 1.0 / norm_temp1;
    if (it == 1) 
      printf("\n   iteration           ||r||                 zeta\n");
    printf("    %5d       %20.14E%20.13f\n", it, rnorm, zeta);

    //---------------------------------------------------------------------
    // Normalize z to obtain x
    //---------------------------------------------------------------------
    for (j = 0; j < lastcol - firstcol + 1; j++) {
      x[j] = norm_temp2 * z[j];
    }
  } // end of main iter inv pow meth

  timer_stop(T_bench);

  //---------------------------------------------------------------------
  // End of timed section
  //---------------------------------------------------------------------

  t = timer_read(T_bench);

  printf("\nComplete...\n");

  epsilon = 1.0e-10;
  err = fabs(zeta - zeta_verify_value) / zeta_verify_value;
  if (err <= epsilon) {
    verified = true;
    printf(" VERIFICATION SUCCESSFUL\n");
    printf(" Zeta is    %20.13E\n", zeta);
    printf(" Error is   %20.13E\n", err);
  } else {
    verified = false;
    printf(" VERIFICATION FAILED\n");
    printf(" Zeta                %20.13E\n", zeta);
    printf(" The correct zeta is %20.13E\n", zeta_verify_value);
  }
  
  printf("\n\nExecution time : %lf seconds\n\n", t);
  
  return 0;
}

// thanks to Nvidia paper
// N. Bell, M. Garland, "Efficient sparse matrix-vector multiplication on cuda",
// Nvidia Technical Report NVR-2008–004 Tech. Rep., 2008
__global__ void mat_vec_mul(int n,
    const int * __restrict__ colidx,
    const int * __restrict__ rowstr,
    const double * __restrict__ a,
    const double * __restrict__ pp,
    double * __restrict__ q) {
  int gid, tid, j, k, k_end, lane;
  __shared__ volatile double sum[256];
  gid = threadIdx.x + blockIdx.x * blockDim.x;
  j = gid >> 5;
  lane = gid & 31;

  tid = threadIdx.x;
  sum[tid] = 0.0;
  if (j >= n) return;

  k_end = rowstr[j+1];
  for (k = rowstr[j] + lane; k < k_end; k += 32) {
    sum[tid] = sum[tid] + a[k]*pp[colidx[k]];
  }
  // in the same warp, no sync needed
  if (lane < 16) sum[tid] += sum[tid + 16];
  if (lane < 8) sum[tid] += sum[tid + 8];
  if (lane < 4) sum[tid] += sum[tid + 4];
  if (lane < 2) sum[tid] += sum[tid + 2];
  if (lane < 1) sum[tid] += sum[tid + 1];
  if (lane == 0) q[j] = sum[tid];
}

__device__ void reduce_block_sum(double sum, double *result) {
  __shared__ double s_sum[1024];
  int i, step;
  i = threadIdx.x;
  s_sum[i] = sum;
  __syncthreads();
  for (step = blockDim.x>>1; step >= 1; step >>= 1) {
    if (i < step) s_sum[i] += s_sum[i+step];
    __syncthreads();
  }
  if (i == 0) result[blockIdx.x] = s_sum[0];
}

__global__ void dot_prod(int n, double *a, double *b, double *result) {
  int i, step;
  double sum = 0.0;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  step = blockDim.x * gridDim.x;
  while (i < n) {
    sum += a[i] * b[i];
    i += step;
  }
  reduce_block_sum(sum, result);
}

__global__ void dist_vec(int n, double *a, double *b, double *result) {
  int i, step;
  double sum = 0.0;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  step = blockDim.x * gridDim.x;
  while (i < n) {
    double d = a[i] - b[i];
    sum += d * d;
    i += step;
  }
  reduce_block_sum(sum, result);
}

__global__ void vec_scale_add(int n, double *a, double scale, double *b, double *result) {
  int i;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= n) return;
  result[i] = a[i] * scale + b[i];
}

//---------------------------------------------------------------------
// Floaging point arrays here are named as in spec discussion of 
// CG algorithm
//---------------------------------------------------------------------
static void conj_grad(int colidx[], // const gpu
                      int rowstr[], // const gpu
                      double x[], // in cpu
                      double z[], // out to cpu
                      double a[], // const gpu
                      double p[], // gpu
                      double q[], // gpu
                      double r[], // gpu
                      double *rnorm)
{
  int j, k;
  int cgit, cgitmax = 25;
  int rowcnt, colcnt;
  int rowblk, colblk;
  double d, sum, rho, rho0, alpha, beta;

  rho = 0.0;
  rowcnt = lastrow - firstrow + 1;
  colcnt = lastcol - firstcol + 1;
  rowblk = (rowcnt + 7) / 8;
  colblk = (colcnt + 255) / 256;

  //---------------------------------------------------------------------
  // Initialize the CG algorithm:
  //---------------------------------------------------------------------
  /*for (j = 0; j < naa+1; j++) {
    q[j] = 0.0;
    z[j] = 0.0;
    r[j] = x[j];
    p[j] = r[j];
  }*/
  hipMemcpy(gpu_x, x, sizeof(double) * (naa+1), hipMemcpyHostToDevice);
  hipMemset(gpu_q, 0, sizeof(double) * (naa+1));
  hipMemset(gpu_z, 0, sizeof(double) * (naa+1));
  hipMemcpy(gpu_r, x, sizeof(double) * (naa+1), hipMemcpyHostToDevice);
  hipMemcpy(gpu_p, x, sizeof(double) * (naa+1), hipMemcpyHostToDevice);

  //---------------------------------------------------------------------
  // rho = r.r
  // Now, obtain the norm of r: First, sum squares of r elements locally...
  //---------------------------------------------------------------------
  /*for (j = 0; j < lastcol - firstcol + 1; j++) {
    rho = rho + r[j]*r[j];
  }*/
  dot_prod<<<colblk/4+1, 256>>>(colcnt, gpu_r, gpu_r, gpu_sum);
  hipMemcpy(cpu_sum, gpu_sum, sizeof(double) * colblk, hipMemcpyDeviceToHost);
  for (j = 0; j < colblk/4+1; j++) {
    rho = rho + cpu_sum[j];
  }

  //---------------------------------------------------------------------
  //---->
  // The conj grad iteration loop
  //---->
  //---------------------------------------------------------------------
  for (cgit = 1; cgit <= cgitmax; cgit++) {
    //---------------------------------------------------------------------
    // q = A.p
    // The partition submatrix-vector multiply: use workspace w
    //---------------------------------------------------------------------
    //
    // NOTE: this version of the multiply is actually (slightly: maybe %5) 
    //       faster on the sp2 on 16 nodes than is the unrolled-by-2 version 
    //       below.   On the Cray t3d, the reverse is true, i.e., the 
    //       unrolled-by-two version is some 10% faster.  
    //       The unrolled-by-8 version below is significantly faster
    //       on the Cray t3d - overall speed of code is 1.5 times faster.

    /*for (j = 0; j < lastrow - firstrow + 1; j++) {
      sum = 0.0;
      for (k = rowstr[j]; k < rowstr[j+1]; k++) {
        sum = sum + a[k]*p[colidx[k]];
      }
      q[j] = sum;
    }*/
    mat_vec_mul<<<rowblk, 256>>>(rowcnt, gpu_colidx, gpu_rowstr, gpu_a, gpu_p, gpu_q);

    //---------------------------------------------------------------------
    // Obtain p.q
    //---------------------------------------------------------------------
    d = 0.0;
    /*for (j = 0; j < lastcol - firstcol + 1; j++) {
      d = d + p[j]*q[j];
    }*/
    dot_prod<<<colblk/4+1, 256>>>(colcnt, gpu_p, gpu_q, gpu_sum);
    hipMemcpy(cpu_sum, gpu_sum, sizeof(double) * colblk, hipMemcpyDeviceToHost);
    for (j = 0; j < colblk/4+1; j++) {
      d = d + cpu_sum[j];
    }

    //---------------------------------------------------------------------
    // Obtain alpha = rho / (p.q)
    //---------------------------------------------------------------------
    alpha = rho / d;

    //---------------------------------------------------------------------
    // Save a temporary of rho
    //---------------------------------------------------------------------
    rho0 = rho;

    //---------------------------------------------------------------------
    // Obtain z = z + alpha*p
    // and    r = r - alpha*q
    //---------------------------------------------------------------------
    rho = 0.0;
    /*for (j = 0; j < lastcol - firstcol + 1; j++) {
      z[j] = z[j] + alpha*p[j];  
      r[j] = r[j] - alpha*q[j];
    }*/
    vec_scale_add<<<colblk, 256>>>(colcnt, gpu_p, alpha, gpu_z, gpu_z);
    vec_scale_add<<<colblk, 256>>>(colcnt, gpu_q, -alpha, gpu_r, gpu_r);
            
    //---------------------------------------------------------------------
    // rho = r.r
    // Now, obtain the norm of r: First, sum squares of r elements locally...
    //---------------------------------------------------------------------
    /*for (j = 0; j < lastcol - firstcol + 1; j++) {
      rho = rho + r[j]*r[j];
    }*/
    dot_prod<<<colblk/4+1, 256>>>(colcnt, gpu_r, gpu_r, gpu_sum);
    hipMemcpy(cpu_sum, gpu_sum, sizeof(double) * colblk, hipMemcpyDeviceToHost);
    for (j = 0; j < colblk/4+1; j++) {
      rho = rho + cpu_sum[j];
    }

    //---------------------------------------------------------------------
    // Obtain beta:
    //---------------------------------------------------------------------
    beta = rho / rho0;

    //---------------------------------------------------------------------
    // p = r + beta*p
    //---------------------------------------------------------------------
    /*for (j = 0; j < lastcol - firstcol + 1; j++) {
      p[j] = r[j] + beta*p[j];
    }*/
    vec_scale_add<<<colblk, 256>>>(colcnt, gpu_p, beta, gpu_r, gpu_p);
  } // end of do cgit=1,cgitmax

  //---------------------------------------------------------------------
  // Compute residual norm explicitly:  ||r|| = ||x - A.z||
  // First, form A.z
  // The partition submatrix-vector multiply
  //---------------------------------------------------------------------
  sum = 0.0;
  /*for (j = 0; j < lastrow - firstrow + 1; j++) {
    d = 0.0;
    for (k = rowstr[j]; k < rowstr[j+1]; k++) {
      d = d + a[k]*z[colidx[k]];
    }
    r[j] = d;
  }*/
  mat_vec_mul<<<rowblk, 256>>>(rowcnt, gpu_colidx, gpu_rowstr, gpu_a, gpu_z, gpu_r);

  //---------------------------------------------------------------------
  // At this point, r contains A.z
  //---------------------------------------------------------------------
  /*for (j = 0; j < lastcol-firstcol+1; j++) {
    d   = x[j] - r[j];
    sum = sum + d*d;
  }*/
  dist_vec<<<colblk/4+1, 256>>>(colcnt, gpu_x, gpu_r, gpu_sum);
  hipMemcpy(cpu_sum, gpu_sum, sizeof(double) * colblk, hipMemcpyDeviceToHost);
  for (j = 0; j < colblk/4+1; j++) {
    sum = sum + cpu_sum[j];
  }

  *rnorm = sqrt(sum);

  hipMemcpy(z, gpu_z, sizeof(double) * colcnt, hipMemcpyDeviceToHost);
}


//---------------------------------------------------------------------
// generate the test problem for benchmark 6
// makea generates a sparse matrix with a
// prescribed sparsity distribution
//
// parameter    type        usage
//
// input
//
// n            i           number of cols/rows of matrix
// nz           i           nonzeros as declared array size
// rcond        r*8         condition number
// shift        r*8         main diagonal shift
//
// output
//
// a            r*8         array for nonzeros
// colidx       i           col indices
// rowstr       i           row pointers
//
// workspace
//
// iv, arow, acol i
// aelt           r*8
//---------------------------------------------------------------------
static void makea(int n,
                  int nz,
                  double a[],
                  int colidx[],
                  int rowstr[],
                  int firstrow,
                  int lastrow,
                  int firstcol,
                  int lastcol,
                  int arow[],
                  int acol[][NONZER+1],
                  double aelt[][NONZER+1],
                  int iv[])
{
  int iouter, ivelt, nzv, nn1;
  int ivc[NONZER+1];
  double vc[NONZER+1];

  //---------------------------------------------------------------------
  // nonzer is approximately  (int(sqrt(nnza /n)));
  //---------------------------------------------------------------------

  //---------------------------------------------------------------------
  // nn1 is the smallest power of two not less than n
  //---------------------------------------------------------------------
  nn1 = 1;
  do {
    nn1 = 2 * nn1;
  } while (nn1 < n);

  //---------------------------------------------------------------------
  // Generate nonzero positions and save for the use in sparse.
  //---------------------------------------------------------------------
  for (iouter = 0; iouter < n; iouter++) {
    nzv = NONZER;
    sprnvc(n, nzv, nn1, vc, ivc);
    vecset(n, vc, ivc, &nzv, iouter+1, 0.5);
    arow[iouter] = nzv;
    
    for (ivelt = 0; ivelt < nzv; ivelt++) {
      acol[iouter][ivelt] = ivc[ivelt] - 1;
      aelt[iouter][ivelt] = vc[ivelt];
    }
  }

  //---------------------------------------------------------------------
  // ... make the sparse matrix from list of elements with duplicates
  //     (iv is used as  workspace)
  //---------------------------------------------------------------------
  sparse(a, colidx, rowstr, n, nz, NONZER, arow, acol, 
         aelt, firstrow, lastrow,
         iv, RCOND, SHIFT);
}


//---------------------------------------------------------------------
// rows range from firstrow to lastrow
// the rowstr pointers are defined for nrows = lastrow-firstrow+1 values
//---------------------------------------------------------------------
static void sparse(double a[],
                   int colidx[],
                   int rowstr[],
                   int n,
                   int nz,
                   int nozer,
                   int arow[],
                   int acol[][NONZER+1],
                   double aelt[][NONZER+1],
                   int firstrow,
                   int lastrow,
                   int nzloc[],
                   double rcond,
                   double shift)
{
  int nrows;

  //---------------------------------------------------
  // generate a sparse matrix from a list of
  // [col, row, element] tri
  //---------------------------------------------------
  int i, j, j1, j2, nza, k, kk, nzrow, jcol;
  double size, scale, ratio, va;
  logical cont40;

  //---------------------------------------------------------------------
  // how many rows of result
  //---------------------------------------------------------------------
  nrows = lastrow - firstrow + 1;

  //---------------------------------------------------------------------
  // ...count the number of triples in each row
  //---------------------------------------------------------------------
  for (j = 0; j < nrows+1; j++) {
    rowstr[j] = 0;
  }

  for (i = 0; i < n; i++) {
    for (nza = 0; nza < arow[i]; nza++) {
      j = acol[i][nza] + 1;
      rowstr[j] = rowstr[j] + arow[i];
    }
  }

  rowstr[0] = 0;
  for (j = 1; j < nrows+1; j++) {
    rowstr[j] = rowstr[j] + rowstr[j-1];
  }
  nza = rowstr[nrows] - 1;

  //---------------------------------------------------------------------
  // ... rowstr(j) now is the location of the first nonzero
  //     of row j of a
  //---------------------------------------------------------------------
  if (nza > nz) {
    printf("Space for matrix elements exceeded in sparse\n");
    printf("nza, nzmax = %d, %d\n", nza, nz);
    exit(EXIT_FAILURE);
  }

  //---------------------------------------------------------------------
  // ... preload data pages
  //---------------------------------------------------------------------
  for (j = 0; j < nrows; j++) {
    for (k = rowstr[j]; k < rowstr[j+1]; k++) {
      a[k] = 0.0;
      colidx[k] = -1;
    }
    nzloc[j] = 0;
  }

  //---------------------------------------------------------------------
  // ... generate actual values by summing duplicates
  //---------------------------------------------------------------------
  size = 1.0;
  ratio = pow(rcond, (1.0 / (double)(n)));

  for (i = 0; i < n; i++) {
    for (nza = 0; nza < arow[i]; nza++) {
      j = acol[i][nza];

      scale = size * aelt[i][nza];
      for (nzrow = 0; nzrow < arow[i]; nzrow++) {
        jcol = acol[i][nzrow];
        va = aelt[i][nzrow] * scale;

        //--------------------------------------------------------------------
        // ... add the identity * rcond to the generated matrix to bound
        //     the smallest eigenvalue from below by rcond
        //--------------------------------------------------------------------
        if (jcol == j && j == i) {
          va = va + rcond - shift;
        }

        cont40 = false;
        for (k = rowstr[j]; k < rowstr[j+1]; k++) {
          if (colidx[k] > jcol) {
            //----------------------------------------------------------------
            // ... insert colidx here orderly
            //----------------------------------------------------------------
            for (kk = rowstr[j+1]-2; kk >= k; kk--) {
              if (colidx[kk] > -1) {
                a[kk+1]  = a[kk];
                colidx[kk+1] = colidx[kk];
              }
            }
            colidx[k] = jcol;
            a[k]  = 0.0;
            cont40 = true;
            break;
          } else if (colidx[k] == -1) {
            colidx[k] = jcol;
            cont40 = true;
            break;
          } else if (colidx[k] == jcol) {
            //--------------------------------------------------------------
            // ... mark the duplicated entry
            //--------------------------------------------------------------
            nzloc[j] = nzloc[j] + 1;
            cont40 = true;
            break;
          }
        }
        if (cont40 == false) {
          printf("internal error in sparse: i=%d\n", i);
          exit(EXIT_FAILURE);
        }
        a[k] = a[k] + va;
      }
    }
    size = size * ratio;
  }

  //---------------------------------------------------------------------
  // ... remove empty entries and generate final results
  //---------------------------------------------------------------------
  for (j = 1; j < nrows; j++) {
    nzloc[j] = nzloc[j] + nzloc[j-1];
  }

  for (j = 0; j < nrows; j++) {
    if (j > 0) {
      j1 = rowstr[j] - nzloc[j-1];
    } else {
      j1 = 0;
    }
    j2 = rowstr[j+1] - nzloc[j];
    nza = rowstr[j];
    for (k = j1; k < j2; k++) {
      a[k] = a[nza];
      colidx[k] = colidx[nza];
      nza = nza + 1;
    }
  }
  for (j = 1; j < nrows+1; j++) {
    rowstr[j] = rowstr[j] - nzloc[j-1];
  }
  nza = rowstr[nrows] - 1;
}


//---------------------------------------------------------------------
// generate a sparse n-vector (v, iv)
// having nzv nonzeros
//
// mark(i) is set to 1 if position i is nonzero.
// mark is all zero on entry and is reset to all zero before exit
// this corrects a performance bug found by John G. Lewis, caused by
// reinitialization of mark on every one of the n calls to sprnvc
//---------------------------------------------------------------------
static void sprnvc(int n, int nz, int nn1, double v[], int iv[])
{
  int nzv, ii, i;
  double vecelt, vecloc;

  nzv = 0;

  while (nzv < nz) {
    vecelt = randlc(&tran, amult);

    //---------------------------------------------------------------------
    // generate an integer between 1 and n in a portable manner
    //---------------------------------------------------------------------
    vecloc = randlc(&tran, amult);
    i = icnvrt(vecloc, nn1) + 1;
    if (i > n) continue;

    //---------------------------------------------------------------------
    // was this integer generated already?
    //---------------------------------------------------------------------
    logical was_gen = false;
    for (ii = 0; ii < nzv; ii++) {
      if (iv[ii] == i) {
        was_gen = true;
        break;
      }
    }
    if (was_gen) continue;
    v[nzv] = vecelt;
    iv[nzv] = i;
    nzv = nzv + 1;
  }
}


//---------------------------------------------------------------------
// scale a double precision number x in (0,1) by a power of 2 and chop it
//---------------------------------------------------------------------
static int icnvrt(double x, int ipwr2)
{
  return (int)(ipwr2 * x);
}


//---------------------------------------------------------------------
// set ith element of sparse vector (v, iv) with
// nzv nonzeros to val
//---------------------------------------------------------------------
static void vecset(int n, double v[], int iv[], int *nzv, int i, double val)
{
  int k;
  logical set;

  set = false;
  for (k = 0; k < *nzv; k++) {
    if (iv[k] == i) {
      v[k] = val;
      set  = true;
    }
  }
  if (set == false) {
    v[*nzv]  = val;
    iv[*nzv] = i;
    *nzv     = *nzv + 1;
  }
}

